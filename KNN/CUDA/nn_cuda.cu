/*
 * nn.cu
 * Nearest Neighbor
 *
 */

#include <stdio.h>
#include <sys/time.h>
#include <float.h>
#include <vector>
#include "hip/hip_runtime.h"

#ifdef TIMING
#include "timing.h"

struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
lat_type init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;
#endif

#define min( a, b )			a > b ? b : a
#define ceilDiv( a, b )		( a + b - 1 ) / b
#define print( x )			printf( #x ": %lu\n", (unsigned long) x )
#define DEBUG				false

#define DEFAULT_THREADS_PER_BLOCK 256

#define MAX_ARGS 10
#define REC_LENGTH 53 // size of a record in db
#define LATITUDE_POS 28	// character position of the latitude value in each record
#define OPEN 10000	// initial value of nearest neighbors

int cpu_offset = 0;

typedef double lat_type;

double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}

typedef struct latLong
{
  lat_type lat;
  lat_type lng;
} LatLong;

typedef struct record
{
  char recString[REC_LENGTH];
  lat_type distance;
} Record;

int loadData(char *filename,std::vector<Record> &records,std::vector<LatLong> &locations);
void findLowest(std::vector<Record> &records,lat_type *distances,int numRecords,int topN);
void printUsage();
int parseCommandline(int argc, char *argv[], char* filename,int *r,lat_type *lat,lat_type *lng,
                     int *q, int *t, int *p, int *d);

/**
* Kernel
* Executed on GPU
* Calculates the Euclidean distance from each record in the database to the target position
*/
__global__ void euclid(LatLong *d_locations, lat_type *d_distances, int numRecords,lat_type lat, lat_type lng)
{
	//int globalId = gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
	int globalId = blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x ) + threadIdx.x; // more efficient
    LatLong *latLong = d_locations+globalId;
    if (globalId < numRecords) {
        lat_type *dist=d_distances+globalId;
        *dist = (lat_type)sqrt((lat-latLong->lat)*(lat-latLong->lat)+(lng-latLong->lng)*(lng-latLong->lng));
	}
}

void euclid_cpu(LatLong *locations,
                        lat_type *distances,
                        const int gpu_numRecords,
                        const int numRecords,
                        const lat_type lat,
                        const lat_type lng)
{
#pragma omp parallel for
    for (int i = gpu_numRecords; i < numRecords; i++)
    {
        LatLong *latLong = locations + i;
        lat_type *dist = distances + i;
        *dist = (lat_type)sqrt((lat - latLong->lat) * (lat - latLong->lat) + (lng - latLong->lng) * (lng - latLong->lng));
    }
}

/**
* This program finds the k-nearest neighbors
**/

int main(int argc, char* argv[])
{
	int    i=0;
	lat_type lat, lng;
	int quiet=0,timing=0,platform=0,device=0;

    std::vector<Record> records;
	std::vector<LatLong> locations;
	char filename[100];
	int resultsCount=10;

    // parse command line
    if (parseCommandline(argc, argv, filename,&resultsCount,&lat,&lng,
                     &quiet, &timing, &platform, &device)) {
      printUsage();
      return 0;
    }

    int numRecords = loadData(filename,records,locations);
    if (resultsCount > numRecords) resultsCount = numRecords;

    //for(i=0;i<numRecords;i++)
    //  printf("%s, %f, %f\n",(records[i].recString),locations[i].lat,locations[i].lng);


    //Pointers to host memory
	lat_type *distances;
	//Pointers to device memory
	LatLong *d_locations;
	lat_type *d_distances;


	// Scaling calculations - added by Sam Kauffman
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties( &deviceProp, 0 );
	hipDeviceSynchronize();
	unsigned long maxGridX = deviceProp.maxGridSize[0];
	unsigned long threadsPerBlock = min( deviceProp.maxThreadsPerBlock, DEFAULT_THREADS_PER_BLOCK );
	size_t totalDeviceMemory;
	size_t freeDeviceMemory;
	hipMemGetInfo(  &freeDeviceMemory, &totalDeviceMemory );
	hipDeviceSynchronize();
	unsigned long usableDeviceMemory = freeDeviceMemory * 85 / 100; // 85% arbitrary throttle to compensate for known CUDA bug
	unsigned long maxThreads = usableDeviceMemory / 12; // 4 bytes in 3 vectors per thread
	if ( numRecords > maxThreads )
	{
		fprintf( stderr, "Error: Input too large.\n" );
		exit( 1 );
	}
	unsigned long blocks = ceilDiv( numRecords, threadsPerBlock ); // extra threads will do nothing
	unsigned long gridY = ceilDiv( blocks, maxGridX );
	unsigned long gridX = ceilDiv( blocks, gridY );
	// There will be no more than (gridY - 1) extra blocks
	dim3 gridDim( gridX, gridY );

	if ( DEBUG )
	{
		print( totalDeviceMemory ); // 804454400
		print( freeDeviceMemory );
		print( usableDeviceMemory );
		print( maxGridX ); // 65535
		print( deviceProp.maxThreadsPerBlock ); // 1024
		print( threadsPerBlock );
		print( maxThreads );
		print( blocks ); // 130933
		print( gridY );
		print( gridX );
	}

	/**
	* Allocate memory on host and device
	*/
	distances = (lat_type *)malloc(sizeof(lat_type) * numRecords);
	hipMalloc((void **) &d_locations,sizeof(LatLong) * numRecords);
	hipMalloc((void **) &d_distances,sizeof(lat_type) * numRecords);

   /**
    * Transfer data from host to device
    */
    hipMemcpy( d_locations, &locations[0], sizeof(LatLong) * numRecords, hipMemcpyHostToDevice);

    /**
    * Execute kernel
    */
    int cpu_numRecords = numRecords * cpu_offset / 100;
    int gpu_numRecords = numRecords - cpu_numRecords;

    double t1 = gettime();
    if (cpu_offset < 100)
        euclid<<< gridDim, threadsPerBlock >>>(d_locations,d_distances,gpu_numRecords,lat,lng);

    if (cpu_offset > 0)
        euclid_cpu(&locations[0], distances, gpu_numRecords, numRecords, lat, lng);

    if (cpu_offset < 100)
        hipDeviceSynchronize();



    //Copy data from device memory to host memory
    if (cpu_offset < 100)
        hipMemcpy( distances, d_distances, sizeof(lat_type)*gpu_numRecords, hipMemcpyDeviceToHost );


	// find the resultsCount least distances
    findLowest(records,distances,numRecords,resultsCount);

    // print out results
    if (!quiet)
    for(i=0;i<resultsCount;i++) {
      printf("%s --> Distance=%f\n",records[i].recString,records[i].distance);
    }
    printf("CPU offset: %d\n", cpu_offset);
    double t2 = gettime();

    printf("Total time: %lf\n\n", (t2-t1));

    free(distances);
    //Free memory
	hipFree(d_locations);
	hipFree(d_distances);

#ifdef  TIMING
    printf("Exec: %f\n", kernel_time);
#endif
}

int loadData(char *filename,std::vector<Record> &records,std::vector<LatLong> &locations){
    FILE   *flist,*fp;
	int    i=0;
	char dbname[64];
	int recNum=0;

    /**Main processing **/

    flist = fopen(filename, "r");
	while(!feof(flist)) {
		/**
		* Read in all records of length REC_LENGTH
		* If this is the last file in the filelist, then done
		* else open next file to be read next iteration
		*/
		if(fscanf(flist, "%s\n", dbname) != 1) {
            fprintf(stderr, "error reading filelist\n");
            exit(0);
        }
        fp = fopen(dbname, "r");
        if(!fp) {
            printf("error opening a db\n");
            exit(1);
        }
        // read each record
        while(!feof(fp)){
            Record record;
            LatLong latLong;
            fgets(record.recString,49,fp);
            fgetc(fp); // newline
            if (feof(fp)) break;

            // parse for lat and long
            char substr[6];

            for(i=0;i<5;i++) substr[i] = *(record.recString+i+28);
            substr[5] = '\0';
            latLong.lat = atof(substr);

            for(i=0;i<5;i++) substr[i] = *(record.recString+i+33);
            substr[5] = '\0';
            latLong.lng = atof(substr);

            locations.push_back(latLong);
            records.push_back(record);
            recNum++;
        }
        fclose(fp);
    }
    fclose(flist);
//    for(i=0;i<rec_count*REC_LENGTH;i++) printf("%c",sandbox[i]);
    return recNum;
}

void findLowest(std::vector<Record> &records,lat_type *distances,int numRecords,int topN){
  int i,j;
  lat_type val;
  int minLoc;
  Record *tempRec;
  lat_type tempDist;

  for(i=0;i<topN;i++) {
    minLoc = i;
    for(j=i;j<numRecords;j++) {
      val = distances[j];
      if (val < distances[minLoc]) minLoc = j;
    }
    // swap locations and distances
    tempRec = &records[i];
    records[i] = records[minLoc];
    records[minLoc] = *tempRec;

    tempDist = distances[i];
    distances[i] = distances[minLoc];
    distances[minLoc] = tempDist;

    // add distance to the min we just found
    records[i].distance = distances[i];
  }
}

int parseCommandline(int argc, char *argv[], char* filename,int *r,lat_type *lat,lat_type *lng,
                     int *q, int *t, int *p, int *d){
    int i;
    if (argc < 2) return 1; // error
    strncpy(filename,argv[1],100);
    char flag;

    for(i=1;i<argc;i++) {
      if (argv[i][0]=='-') {// flag
        flag = argv[i][1];
          switch (flag) {
            case 'o':
              i++;
              cpu_offset = atoi(argv[i]);
              break;
            case 'r': // number of results
              i++;
              *r = atoi(argv[i]);
              break;
            case 'l': // lat or lng
              if (argv[i][2]=='a') {//lat
                *lat = atof(argv[i+1]);
              }
              else {//lng
                *lng = atof(argv[i+1]);
              }
              i++;
              break;
            case 'h': // help
              return 1;
            case 'q': // quiet
              *q = 1;
              break;
            case 't': // timing
              *t = 1;
              break;
            case 'p': // platform
              i++;
              *p = atoi(argv[i]);
              break;
            case 'd': // device
              i++;
              *d = atoi(argv[i]);
              break;
        }
      }
    }
    if ((*d >= 0 && *p<0) || (*p>=0 && *d<0)) // both p and d must be specified if either are specified
      return 1;
    return 0;
}

void printUsage(){
  printf("Nearest Neighbor Usage\n");
  printf("\n");
  printf("nearestNeighbor [filename] -r [int] -lat [lat_type] -lng [lat_type] [-hqt] [-p [int] -d [int]]\n");
  printf("\n");
  printf("example:\n");
  printf("$ ./nearestNeighbor filelist.txt -r 5 -lat 30 -lng 90\n");
  printf("\n");
  printf("filename     the filename that lists the data input files\n");
  printf("-r [int]     the number of records to return (default: 10)\n");
  printf("-lat [lat_type] the latitude for nearest neighbors (default: 0)\n");
  printf("-lng [lat_type] the longitude for nearest neighbors (default: 0)\n");
  printf("\n");
  printf("-h, --help   Display the help file\n");
  printf("-q           Quiet mode. Suppress all text output.\n");
  printf("-t           Print timing information.\n");
  printf("\n");
  printf("-p [int]     Choose the platform (must choose both platform and device)\n");
  printf("-d [int]     Choose the device (must choose both platform and device)\n");
  printf("\n");
  printf("\n");
  printf("Notes: 1. The filename is required as the first parameter.\n");
  printf("       2. If you declare either the device or the platform,\n");
  printf("          you must declare both.\n\n");
}